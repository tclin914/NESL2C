#include "hip/hip_runtime.h"
#include <stdio.h>
#include "pf/pf.h"
struct tuple_F_F
{
float a;
float b;
};

struct tuple_tuple_F_F_tuple_F_F
{
struct tuple_F_F a;
struct tuple_F_F b;
};

struct tuple_F_F;

struct tuple_tuple_F_F_tuple_tuple_F_F_tuple_F_F
{
struct tuple_F_F a;
struct tuple_tuple_F_F_tuple_F_F b;
};

struct tuple_tuple_F_F_tuple_F_F;

float __device__ __forceinline__ cross_product(struct tuple_tuple_F_F_tuple_tuple_F_F_tuple_F_F tmp1)
{
float res;
struct tuple_F_F tmp2;
struct tuple_tuple_F_F_tuple_F_F tmp3;
float t1, t2, t3, t4, t5, t6, t7;
tmp2=tmp1.a;
tmp3=tmp1.b;
t1=(tmp3.a.a-tmp2.a);
t2=(tmp3.b.b-tmp2.b);
t3=(t1*t2);
t4=(tmp3.a.b-tmp2.a);
t5=(tmp3.b.a-tmp2.b);
t6=(t4*t5);
t7=(t3-t6);
res=t7;
return res;
}

void __device__ myFunc1_deviceEntry00(int32_t mask00)
{
struct tuple_F_F tmp4, tmp5, tmp6;
struct tuple_tuple_F_F_tuple_F_F tmp7;
struct tuple_tuple_F_F_tuple_tuple_F_F_tuple_F_F tmp8;
float res;
if ((mask00&1)==1)
{
tmp4.a=1.0;
tmp4.b=1.0;
tmp5.a=5.0;
tmp5.b=2.4;
tmp6.a=4.1;
tmp6.b=4.3;
tmp7.a=tmp5;
tmp7.b=tmp6;
tmp8.a=tmp4;
tmp8.b=tmp7;
res=cross_product(tmp8);
printf("res = %f\n", res);
}
}

void __global__ __launch_bounds__(BLOCK_SIZE, (NUM_BLOCKS/NUM_SMS)) myFunc1()
{
bool mask00;
initializeContext();
mask00=((gangID()==0)&&(workerID()==0));
if (anyTrue(mask00)||getWorkAndSync(0, 0))
{
incRecurLevel(0);
myFunc1_deviceEntry00((mask00 ? 1 : 0));
getWorkAndSync(0, 0);
}
}

int main()
{
CHECK_ERR(hipDeviceSetLimit(hipLimitStackSize, LOCAL_STACK_SIZE));
CHECK_ERR(hipFuncSetCacheConfig(reinterpret_cast<const void*>(myFunc1), hipFuncCachePreferShared));
initializeGPUData();
myFunc1<<<NUM_BLOCKS, BLOCK_SIZE>>>();
hipDeviceSynchronize();
return 1;
}

